#include "hip/hip_runtime.h"
/* dkoes
 * This file contains all the standalone gpu kernels.  There is (hopefully)
 * a nicer way to organize this, but I'm currently slightly flummoxed as to
 * how to cleaning mix object-oriented cpu and gpu code.
 */
#include "gpucode.h"
#include <thrust/reduce.h>
#include <thrust/device_ptr.h>
#include <stdio.h>

__global__ void evaluate_splines(float **splines, float r, float fraction,
		float cutoff, float *vals, float *derivs)
{
	unsigned i = blockIdx.x;
	float *spline = splines[i];
	vals[i] = 0;
	derivs[i] = 0;

	if (r >= cutoff || r < 0)
	{
		return;
	}

	unsigned index = r / fraction; //xval*numpoints/cutoff
	unsigned base = 5 * index;
	float x = spline[base];
	float a = spline[base + 1];
	float b = spline[base + 2];
	float c = spline[base + 3];
	float d = spline[base + 4];

	const float lx = r - x;
	vals[i] = ((a * lx + b) * lx + c) * lx + d;
	derivs[i] = (3 * a * lx + 2 * b) * lx + c;
}

//TODO: buy compute 3.0 or greater card and implement dynamic paralellism
//evaluate a single spline
__device__ float evaluate_spline(float *spline, float r, float fraction,
		float cutoff, float& deriv)
{
	float val = 0;
	deriv = 0;
	if (r >= cutoff || r < 0)
	{
		return 0;
	}

	unsigned index = r / fraction; //xval*numpoints/cutoff
	unsigned base = 5 * index;
	float x = spline[base];
	float a = spline[base + 1];
	float b = spline[base + 2];
	float c = spline[base + 3];
	float d = spline[base + 4];

	const float lx = r - x;
	val = ((a * lx + b) * lx + c) * lx + d;
	deriv = (3 * a * lx + 2 * b) * lx + c;
	return val;
}

void evaluate_splines_host(const GPUSplineInfo& spInfo,
		float r, std::vector<float>& vals, std::vector<float>& derivs)
{
	unsigned n = spInfo.n;
	vals.resize(n);
	derivs.resize(n);

	float *device_vals, *device_derivs;
	hipMalloc(&device_vals, sizeof(float) * n);
	hipMalloc(&device_derivs, sizeof(float) * n);

	evaluate_splines<<<n,1>>>((float**)spInfo.splines, r, spInfo.fraction, spInfo.cutoff,
			device_vals, device_derivs);

	hipMemcpy(&vals[0], device_vals, n * sizeof(float),
			hipMemcpyDeviceToHost);
	hipMemcpy(&derivs[0], device_derivs, n * sizeof(float),
			hipMemcpyDeviceToHost);
	hipFree(device_vals);
	hipFree(device_derivs);

}

__device__ float eval_deriv_gpu(GPUNonCacheInfo *dinfo, unsigned t,
		float charge, unsigned rt, float rcharge, float r2, float& dor)
{
	float r = sqrt(r2);
	unsigned t1, t2;
	float charge1, charge2;
	if (t < rt)
	{
		t1 = t;
		t2 = rt;
		charge1 = fabs(charge);
		charge2 = fabs(rcharge);
	}
	else
	{
		t1 = rt;
		t2 = t;
		charge1 = fabs(rcharge);
		charge2 = fabs(charge);
	}

	unsigned tindex = t1 + t2 * (t2 + 1) / 2;
	GPUSplineInfo spInfo = dinfo->splineInfo[tindex];
	unsigned n = spInfo.n; //number of components

	float ret = 0, d = 0;

	//ick, hard code knowledge of components here; need to come up with
	//something mroe elegant
	//TypeDependentOnly,//no need to adjust by charge
	if (n > 0)
	{
		float fraction = spInfo.fraction;
		float cutoff = spInfo.cutoff;
		float val, deriv;
		val = evaluate_spline(spInfo.splines[0], r, fraction, cutoff, deriv);
		ret += val;
		d += deriv;
		//AbsAChargeDependent,//multiply by the absolute value of a's charge
		if (n > 1)
		{
			val = evaluate_spline(spInfo.splines[1], r, fraction, cutoff,
					deriv);
			ret += val * charge1;
			d += deriv * charge1;
			//AbsBChargeDependent,//multiply by abs(b)'s charge
			if (n > 2)
			{
				val = evaluate_spline(spInfo.splines[2], r, fraction, cutoff,
						deriv);
				ret += val * charge2;
				d += deriv * charge2;
				//ABChargeDependent,//multiply by a*b
				if (n > 3)
				{
					val = evaluate_spline(spInfo.splines[3], r, fraction,
							cutoff, deriv);
					ret += val * charge2 * charge1;
					d += deriv * charge2 * charge1;
				}
			}
		}
	}

	dor = d / r; //divide by distance to normalize vector later
	return ret;
}

//curl function to scale back positive energies and match vina calculations
//assume v is reasonable
__device__ void curl(float& e, float *deriv, float v)
{
	if (e > 0)
	{
		float tmp = (v / (v + e));
		e *= tmp;
		tmp *= tmp;
		for (unsigned i = 0; i < 3; i++)
			deriv[i] *= tmp;
	}
}

//calculates the energies of all ligand-prot interactions and combines the results
//into energies and minus forces
//needs enough shared memory for derivatives and energies of single ligand atom
//roffset specifies how far into the receptor atoms we are
__global__ void interaction_energy(GPUNonCacheInfo *dinfo, unsigned roffset,
		float slope, float v)
{
	unsigned l = blockIdx.x;
	unsigned r = threadIdx.x;
	unsigned ridx = roffset + r;
	//get ligand atom info
	unsigned t = dinfo->types[l];
	//TODO: remove hydrogen atoms completely
	if (t <= 1) //hydrogen ligand atom
		return;
	float out_of_bounds_deriv[3] =
			{ 0, 0, 0 };
	float out_of_bounds_penalty = 0;

	extern __shared__ float mysmem[];
	float *myenergies = mysmem;
	float *derivs = mysmem+blockDim.x;

	//initailize shared memory
	myenergies[r] = 0;
	derivs[3 * r] = 0;
	derivs[3 * r + 1] = 0;
	derivs[3 * r + 2] = 0;

	float charge = dinfo->charges[l];
	float xyz[3] =
			{ dinfo->coords[3 * l], dinfo->coords[3 * l + 1],
					dinfo->coords[3 * l + 2] };

	//evaluate for out of boundsness
	for (unsigned i = 0; i < 3; i++)
	{
		float min = dinfo->gridbegins[i];
		float max = dinfo->gridends[i];
		if (xyz[i] < min)
		{
			out_of_bounds_deriv[i] = -1;
			out_of_bounds_penalty += fabs(min - xyz[i]);
			xyz[i] = min;
		}
		else if (xyz[i] > max)
		{
			out_of_bounds_deriv[i] = 1;
			out_of_bounds_penalty += fabs(max - xyz[i]);
			xyz[i] = max;
		}
		out_of_bounds_deriv[i] *= slope;
	}

	out_of_bounds_penalty *= slope;

	//now consider interaction with every possible receptor atom
	//TODO: parallelize

	float cutoff = dinfo->cutoff_sq;

	unsigned rt;
	float rcharge;
	float rxyz[3];
	float diff[3];

	rt = dinfo->rectypes[ridx];
	rcharge = dinfo->reccharges[ridx];
	rxyz[0] = dinfo->recoords[3 * ridx];
	rxyz[1] = dinfo->recoords[3 * ridx + 1];
	rxyz[2] = dinfo->recoords[3 * ridx + 2];

	//compute squared difference
	float rSq = 0;
	for (unsigned j = 0; j < 3; j++)
	{
		float d = xyz[j] - rxyz[j];
		diff[j] = d;
		rSq += d * d;
	}

	if (rSq < cutoff)
	{
		//dkoes - the "derivative" value returned by eval_deriv
		//is normalized by r (dor = derivative over r?)
		float dor;
		myenergies[r] = eval_deriv_gpu(dinfo, t, charge, rt, rcharge, rSq,
				dor);
		for (unsigned j = 0; j < 3; j++)
		{
			derivs[3 * r + j] = dor * diff[j];
		}
	}

	__syncthreads();
	//horribly inefficient reduction; TODO improve
	if (r == 0)
	{
		float this_e = 0;
		float deriv[3] =
				{ 0, 0, 0 };
		unsigned nr = blockDim.x;
		for (unsigned i = 0; i < nr; i++)
		{
			this_e += myenergies[i];
			deriv[0] += derivs[3 * i];
			deriv[1] += derivs[3 * i + 1];
			deriv[2] += derivs[3 * i + 2];
		}
		curl(this_e, deriv, v);
		
		//update minus forces
		for (unsigned j = 0; j < 3; j++)
		{
			dinfo->minus_forces[3 * l + j] += deriv[j] + out_of_bounds_deriv[j];
		}
		//and energy
		dinfo->energies[l] += this_e + out_of_bounds_penalty;
	}
}
//calculates the energies of a single ligand atom (determined by block id)
__global__ void per_ligand_atom_energy(GPUNonCacheInfo *dinfo, float slope, float v)
{
        unsigned l = blockIdx.x;

        //get ligand atom info
        unsigned t = dinfo->types[l];

        //TODO: remove hydrogen atoms completely
        if (t <= 1)
                return; // hydrogen

        float charge = dinfo->charges[l];
        float xyz[3] =
        { dinfo->coords[3 * l], dinfo->coords[3 * l + 1],
                        dinfo->coords[3 * l + 2] };

        float out_of_bounds_deriv[3] =
        { 0, 0, 0 };
        float out_of_bounds_penalty = 0;

        //evaluate for out of boundsness
        for (unsigned i = 0; i < 3; i++)
        {
                float min = dinfo->gridbegins[i];
                float max = dinfo->gridends[i];
                if (xyz[i] < min)
                {
                        out_of_bounds_deriv[i] = -1;
                        out_of_bounds_penalty += fabs(min - xyz[i]);
                        xyz[i] = min;
                }
                else if (xyz[i] > max)
                {
                        out_of_bounds_deriv[i] = 1;
                        out_of_bounds_penalty += fabs(max - xyz[i]);
                        xyz[i] = max;
                }
                out_of_bounds_deriv[i] *= slope;
        }

        out_of_bounds_penalty *= slope;

        //now consider interaction with every possible receptor atom
        //TODO: parallelize
        
        float cutoff = dinfo->cutoff_sq;
        float this_e = 0;
        float deriv[3] = {0,0,0};
        unsigned n = dinfo->nrecatoms;
        unsigned rt;
        float rcharge;
        float rxyz[3];
        float diff[3];
        for(unsigned r = 0; r < n; r++) {
                rt = dinfo->rectypes[r];
                rcharge = dinfo->reccharges[r];
                rxyz[0] = dinfo->recoords[3*r];
                rxyz[1] = dinfo->recoords[3*r+1];
                rxyz[2] = dinfo->recoords[3*r+2];
                
                //compute squared difference
                float rSq = 0;
                for(unsigned j = 0; j < 3; j++) {
                        float d = xyz[j]-rxyz[j];
                        diff[j] = d;
                        rSq += d*d;
                }

                if(rSq < cutoff)
                {
                        //dkoes - the "derivative" value returned by eval_deriv
                        //is normalized by r (dor = derivative over r?)
                        float dor;
                        float e = eval_deriv_gpu(dinfo, t, charge, rt, rcharge, rSq, dor);
                        this_e += e;
                        for(unsigned j = 0; j < 3; j++) {
                                deriv[j] +=  dor * diff[j];
                        }
                }
        }

        curl(this_e, deriv, v);
        //update minus forces
        for(unsigned j = 0; j < 3; j++) {
                dinfo->minus_forces[3*l+j] = deriv[j]+out_of_bounds_deriv[j];
        }
        //and energy
        dinfo->energies[l] = this_e + out_of_bounds_penalty;
}


//host side of single point_calculation, energies and coords should already be initialized
float single_point_calc(GPUNonCacheInfo *dinfo, float *energies,
		float slope,
		unsigned natoms, unsigned nrecatoms, float v)
{
#if 1
	//this will calculate the per-atom energies and forces
#define THREADS_PER_BLOCK 1024
	for (unsigned off = 0; off < nrecatoms; off += THREADS_PER_BLOCK)
	{
		unsigned nr = nrecatoms - off;
		if (nr > THREADS_PER_BLOCK)
			nr = THREADS_PER_BLOCK;
		interaction_energy<<<natoms,nr, sizeof(float)*nr*4>>>(dinfo, off,slope, v);
		hipError_t err = hipGetLastError();
		if (hipSuccess != err)
		{
			fprintf(stderr, "cudaCheckError() failed at %s:%i : %s\n",
					__FILE__, __LINE__, hipGetErrorString(err));
			exit(-1);
		}
		hipDeviceSynchronize();
	}
#else
    //this will calculate the per-atom energies and forces
    per_ligand_atom_energy<<<natoms,1>>>(dinfo, slope, v);
#endif
	//get total energy
	thrust::device_ptr<float> dptr(energies);
	return thrust::reduce(dptr, dptr + natoms);
}